
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cudnn_graph.h>
#include <stdio.h>

/******************************************************************************/
/*                                cuda kernels                                */
/******************************************************************************/

/*
 * output = weights * input + biases
 * TODO: use the x axis to compute over the batch
 */
template <int BLOCK_SIZE, typename DataType>
__global__ void
_hhlpLinearForward(DataType const *weights, DataType const *biases,
                   DataType const *inputs, DataType *outputs, int nb_inputs,
                   int nb_outputs, int batch_size) {
    int batch_idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int block_idx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int output_idx = batch_idx * nb_outputs + block_idx;

    if (batch_idx >= batch_size || block_idx >= nb_outputs)
        return;

    DataType result = 0;
    DataType const *weights_row = &weights[block_idx * nb_inputs];
    DataType const *input = &inputs[batch_idx * nb_inputs];
    for (int i = 0; i < nb_inputs; ++i) {
        result += weights_row[i] * input[i];
    }
    outputs[output_idx] = result + biases[block_idx];
}

/*
 * biases_gradient = output_gradient
 */
template <int BLOCK_SIZE, typename DataType>
__global__ void _hhlpLinearBackwardBias(DataType const *output_gradient,
                                        DataType *biases_gradient,
                                        int nb_outputs, int batch_size) {
    int block_idx = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (block_idx >= nb_outputs)
        return;

    DataType result = 0;
    for (int b = 0; b < batch_size; ++b) {
        result += output_gradient[b * nb_outputs + block_idx];
    }
    biases_gradient[block_idx] = result / (DataType)batch_size;
}

/*
 * weights_gradient = output_gradient * inputT
 */
template <int BLOCK_SIZE, typename DataType>
__global__ void
_hhlpLinearBackwardWeights(DataType const *output_gradient,
                           DataType const *input, DataType *weights_gradient,
                           int nb_outputs, int nb_inputs, int batch_size) {
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (row >= nb_outputs || col >= nb_inputs)
        return;

    DataType result = 0;
    for (int b = 0; b < batch_size; ++b) {
        result +=
            output_gradient[b * nb_outputs + row] * input[b * nb_inputs + col];
    }
    weights_gradient[row * nb_inputs + col] = result / (DataType)batch_size;
}

/*
 * input_gradientT = output_gradientT * weights
 */
template <int BLOCK_SIZE, typename DataType>
__global__ void
_hhlpLinearBackwardData(DataType const *output_gradients,
                        DataType const *weights, DataType *input_gradient,
                        int nb_outputs, int nb_inputs, int batch_size) {
    int batch_idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int block_idx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int output_idx = batch_idx * nb_inputs + block_idx;

    if (batch_idx >= batch_size || block_idx >= nb_inputs)
        return;

    DataType result = 0;
    DataType const *weights_col = &weights[block_idx];
    DataType const *output_gradient = &output_gradients[batch_idx * nb_outputs];
    for (int i = 0; i < nb_outputs; ++i) {
        result += output_gradient[i] * weights_col[i * nb_inputs];
    }

    input_gradient[output_idx] = result;
}

/******************************************************************************/
/*                             external functions                             */
/******************************************************************************/

#define EXTERNAL_FUNCTION_IMPL(KERNEL, ...)                                    \
    cudnnStatus_t KERNEL(__VA_ARGS__) {                                        \
        hipStream_t stream;                                                   \
        dim3 threads(1, std::min(nb_outputs, 32));                             \
        dim3 grid(1, std::max(1, nb_outputs / 32));                            \
                                                                               \
        cudnnGetStream(cudnn_handle, &stream);                                 \
        switch (data_type) {                                                   \
        case CUDNN_DATA_FLOAT:                                                 \
            KERNEL<32><<<grid, threads, 0, stream>>>(KERNEL_PARAMS(float));    \
            break;                                                             \
        case CUDNN_DATA_DOUBLE:                                                \
            KERNEL<32><<<grid, threads, 0, stream>>>(KERNEL_PARAMS(double));   \
            break;                                                             \
        case CUDNN_DATA_HALF:                                                  \
            KERNEL<32><<<grid, threads, 0, stream>>>(KERNEL_PARAMS(int16_t));  \
            break;                                                             \
        default:                                                               \
            return cudnnStatus_t::CUDNN_STATUS_NOT_SUPPORTED_DATA_TYPE;        \
            break;                                                             \
        }                                                                      \
        return cudnnStatus_t::CUDNN_STATUS_SUCCESS;                            \
    }

#define LAUNCH_KERNEL(target_type, kernel)                                     \
    {                                                                          \
        using type = target_type;                                              \
        kernel;                                                                \
    }
#define SWITCH_CUDNN_TYPE(data_type, kernel)                                   \
    switch (data_type) {                                                       \
    case CUDNN_DATA_FLOAT:                                                     \
        LAUNCH_KERNEL(float, kernel)                                           \
        break;                                                                 \
    case CUDNN_DATA_DOUBLE:                                                    \
        LAUNCH_KERNEL(double, kernel)                                          \
        break;                                                                 \
    case CUDNN_DATA_HALF:                                                      \
        LAUNCH_KERNEL(__half, kernel)                                          \
        break;                                                                 \
    default:                                                                   \
        return cudnnStatus_t::CUDNN_STATUS_NOT_SUPPORTED_DATA_TYPE;            \
        break;                                                                 \
    }

// TODO: it would be better to take tensor descriptor as argument instead
cudnnStatus_t hhlpLinearForward(cudnnHandle_t cudnn_handle, void const *weights,
                                void const *biases, void const *input,
                                void *output, int nb_inputs, int nb_outputs,
                                int batch_size, cudnnDataType_t data_type) {
    hipStream_t stream;
    cudnnGetStream(cudnn_handle, &stream);

    dim3 threads(32, 32);
    dim3 grid(std::max<int>(1, batch_size / threads.x),
              std::max<int>(1, nb_outputs / threads.y));

    SWITCH_CUDNN_TYPE(
        data_type,
        (_hhlpLinearForward<32><<<grid, threads, 0, stream>>>(
            (type const *)weights, (type const *)biases, (type const *)input,
            (type *)output, nb_inputs, nb_outputs, batch_size)));
    return cudnnStatus_t::CUDNN_STATUS_SUCCESS;
}

cudnnStatus_t hhlpLinearBackwardBias(cudnnHandle_t cudnn_handle,
                                     void const *error, void *biases_gradient,
                                     int nb_outputs, int batch_size,
                                     cudnnDataType_t data_type) {
    hipStream_t stream;
    cudnnGetStream(cudnn_handle, &stream);

    dim3 threads(1, 32);
    dim3 grid(1, (nb_outputs + threads.y - 1) / threads.y);

    SWITCH_CUDNN_TYPE(
        data_type, (_hhlpLinearBackwardBias<32><<<grid, threads, 0, stream>>>(
                       (type const *)error, (type *)biases_gradient, nb_outputs,
                       batch_size)));
    return cudnnStatus_t::CUDNN_STATUS_SUCCESS;
}

cudnnStatus_t hhlpLinearBackwardWeights(cudnnHandle_t cudnn_handle,
                                        void const *output_gradient,
                                        void const *input,
                                        void *weights_gradient, int nb_outputs,
                                        int nb_inputs, int batch_size,
                                        cudnnDataType_t data_type) {
    hipStream_t stream;
    cudnnGetStream(cudnn_handle, &stream);

    dim3 threads(32, 32);
    dim3 grid((nb_inputs + threads.x - 1) / threads.x,
              (nb_outputs + threads.y - 1) / threads.y);

    SWITCH_CUDNN_TYPE(
        data_type,
        (_hhlpLinearBackwardWeights<4><<<grid, threads, 0, stream>>>(
            (type *)output_gradient, (type *)input, (type *)weights_gradient,
            nb_outputs, nb_inputs, batch_size)));
    return cudnnStatus_t::CUDNN_STATUS_SUCCESS;
}

cudnnStatus_t hhlpLinearBackwardData(cudnnHandle_t cudnn_handle,
                                     void const *output_gradient,
                                     void const *weights, void *input_gradient,
                                     int nb_outputs, int nb_inputs,
                                     int batch_size,
                                     cudnnDataType_t data_type) {
    hipStream_t stream;
    cudnnGetStream(cudnn_handle, &stream);

    dim3 threads(32, 32);
    dim3 grid(std::max<int>(1, batch_size / threads.x),
              std::max<int>(1, nb_inputs / threads.y));

    SWITCH_CUDNN_TYPE(
        data_type,
        (_hhlpLinearBackwardData<32><<<grid, threads, 0, stream>>>(
            (type *)output_gradient, (type *)weights, (type *)input_gradient,
            nb_outputs, nb_inputs, batch_size)));
    return cudnnStatus_t::CUDNN_STATUS_SUCCESS;
}
